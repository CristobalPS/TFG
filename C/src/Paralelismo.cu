#include "hip/hip_runtime.h"
/***************************************************************************/
// CRISTÓBAL PÉREZ SIMÓN
// Implementación de las funciones de paralelismo CUDA
/***************************************************************************/
#include "Paralelismo.h"
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

/*************************************************************************/
/*************************************************************************/
// Función de evaluación en la GPU (uso de la librería CUDA)

__global__ void cuda_evaluacion(int * capacidades_device, int c_tamf, 
    int c_tamc, float * fitness_device, int num_estaciones, 
    int * deltas_device, float factor_stress, int porSeguridad, 
    int totalBicis, int * capacidades_total_device, float valor_kms, 
    int * tendencias_device, /*float * tendenciasIn_device, float * tendenciasOut_device, */
    int * usuariosPerdidos_device, int factor_sobrante,
    int * cercanias_device, float * cercaKm_device, int ckm_tamc,
    float * costeKmsExtra_device, float * costeKmsTendencia_device, 
    float * plazasSobrantes_device, 
    int * estaciones_llenas_device, int * estaciones_vacias_device, 
    int d_tamf, int d_tamc, int idest_size, int cerc_tamc, int tend_tamc){
    
    // primero hay que asignar a cada hebra qué individuo evalua
    // segundo comprobar que todos los datos necesarios están disponibles
    // tercero asignar fitness calculado por cada hebra al vector  de resultados
    
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    if(i < c_tamf){
        bool entrada_virt = false;
        bool mov_virtual;
        int contadorKmExtra = 0;
        int llenas = 0;
        int vacias = 0;
        float movido;

        const int n_estaciones = 259;
        int ocupacion[n_estaciones];
        float costeKmsExtra;

        for(int cont = 0; cont < n_estaciones; cont++)
            ocupacion[cont] = 0;
        
        for(int linea_delta = 0; linea_delta < d_tamf; linea_delta++){
            for(int est_Origen = 0; est_Origen < idest_size; est_Origen++){
                // if(i == 0){
                //     printf("iteracion del bucle grande: %d, iteracion del bucle pequeño: %d\n", linea_delta, est_Origen);
                // }
                float pendiente = deltas_device[linea_delta * d_tamc + est_Origen] * factor_stress;
                int nuevatendencia = tendencias_device[linea_delta * tend_tamc + est_Origen];
                int it_estacion = 0;
                int idest = cercanias_device[est_Origen * cerc_tamc];

                if(pendiente == 0){
                    if((ocupacion[est_Origen] == capacidades_device[i * c_tamc + est_Origen] || ocupacion[est_Origen] == 0)){
                        mov_virtual = true;
                        pendiente = nuevatendencia;
                    }
                    else
                        mov_virtual = false;
                }
                else
                    mov_virtual = false;

                if(pendiente > (capacidades_device[i * c_tamc + est_Origen] - ocupacion[est_Origen]))
                    usuariosPerdidos_device[i] += pendiente - (capacidades_device[i * c_tamc + est_Origen] - ocupacion[est_Origen]);
                else if((ocupacion[est_Origen] + pendiente) < 0)
                    usuariosPerdidos_device[i] -= pendiente + ocupacion[est_Origen];

                while(pendiente != 0){
                    if(pendiente > 0){ // Vamos a meter est_Origen
                        if((ocupacion[idest] + pendiente) <= capacidades_device[i * c_tamc + idest]){ // Hay huecos suficientes
                            movido = pendiente;
                            pendiente = 0;
                        }
                        else{ // No hay bastantes huecos
                            movido = capacidades_device[i * c_tamc + idest] - ocupacion[idest];
                            pendiente -= movido;
                        }
                        if(!mov_virtual)
                            ocupacion[idest] = ocupacion[idest] + movido;
                        else{
                            entrada_virt = true;
                            if(ocupacion[idest] == capacidades_device[i * c_tamc + idest])
                                llenas++;
                        }
                    }
                    else{   // Vamos a sacar est_Origen
                        if((ocupacion[idest] + pendiente) >= 0){ // Hay suficientes
                            movido = -pendiente;
                            pendiente = 0;
                        }
                        else{ // Faltan bicis
                            movido = ocupacion[idest];
                            pendiente += movido;
                        }
                        if(!mov_virtual)
                            ocupacion[idest] -= movido;
                        else{
                            entrada_virt = false;
                            if(ocupacion[idest] == 0)
                                vacias++;
                        }
                    }

                    costeKmsExtra = movido * cercaKm_device[est_Origen * ckm_tamc + it_estacion];

                    if(mov_virtual){
                        costeKmsTendencia_device[i] += costeKmsExtra;
                    }
                    else{
                        costeKmsExtra_device[i] += costeKmsExtra;
                        contadorKmExtra++;
                    }
                    
                    // vamos a la siguiente
                    it_estacion++;

                    // si no quedan huecos en ninguna estación
                    if(it_estacion == num_estaciones)
                        break;

                    idest = cercanias_device[est_Origen * cerc_tamc + it_estacion];
            
                }
            }
        }
        float bicis_total = (totalBicis * (1 + porSeguridad) * 1.0);
        plazasSobrantes_device[i] = (capacidades_total_device[i] - bicis_total) / bicis_total;
        fitness_device[i] = ((costeKmsExtra_device[i] + costeKmsTendencia_device[i]) * valor_kms + plazasSobrantes_device[i] * factor_sobrante);
        estaciones_llenas_device[i] = llenas;
        estaciones_vacias_device[i] = vacias;
    }
}

/*************************************************************************/
/*************************************************************************/
// Función de evaluación 
// Recibe:
//      Poblacion: vector con los individuos a evaluar
//      config: datos de configuración
//      deltas_device: matriz con los deltas de las estaciones
//      cercanias_device: matriz con las cercanías de las estaciones
//      cercanias_km_device: matriz con los km de las cercanías de las estaciones
//      tendencias_device: matriz con las tendencias de las estaciones

void evaluarPoblacion_CUDA(vector<Individuo_POO> & Poblacion, Datos_EVAL & config, 
                            int * deltas_device, int * cercanias_device,
                            float * cercanias_km_device, int * tendencias_device,
                            int filas_deltas, int columnas_deltas, int columnas_cercanias_km,
                            int columnas_cercanias, int columnas_tendencias){
    
    vector<float> fitness(Poblacion.size(), 0);
    vector<int> usuariosPerdidos(Poblacion.size(), 0);
    vector<float> costeKmsExtra(Poblacion.size(), 0);
    vector<float> costeKmsTendencia(Poblacion.size(), 0);
    vector<float> plazasSobrantes(Poblacion.size(), 0);
    vector<int> estaciones_llenas(Poblacion.size(), 0);
    vector<int> estaciones_vacias(Poblacion.size(), 0);
    
    try{
        vector<vector<int>> capacidades;
        vector<int> capacidades_total;

        for(int i = 0; i < Poblacion.size(); i++){
            capacidades.push_back(Poblacion[i].getCapacidad());
            capacidades_total.push_back(Poblacion[i].getCapacidadTotal());
            usuariosPerdidos.push_back(Poblacion[i].getUsuariosPerdidos());
        }

        //vector<float> tendenciasIn(Poblacion.size(), 0);
        //vector<float> tendenciasOut(Poblacion.size(), 0);

        // Paso de variables a la GPU
        // CAPACIDADES (pasamos de matriz a vector)
        int filas_capacidades = capacidades.size();
        int columnas_capacidades = capacidades[0].size();
        vector<int> capacidades_vector;

        for(const auto & i : capacidades)
            capacidades_vector.insert(capacidades_vector.end(), i.begin(), i.end());

        int * capacidades_device; // Vector en la GPU
        int capacidades_tam = capacidades_vector.size() * sizeof(int);
        hipMalloc(&capacidades_device, capacidades_tam);
        hipMemcpy(capacidades_device, capacidades_vector.data(), capacidades_tam, hipMemcpyHostToDevice);

        // FITNESS
        float * fitness_device;
        float fitness_tam = fitness.size() * sizeof(float);
        hipMalloc(&fitness_device, fitness_tam);
        hipMemcpy(fitness_device, fitness.data(), fitness_tam, hipMemcpyHostToDevice);

        // CAPACIDADES TOTALES
        int * capacidades_total_device;
        int capacidades_total_tam = capacidades_total.size() * sizeof(int);
        hipMalloc(&capacidades_total_device, capacidades_total_tam);
        hipMemcpy(capacidades_total_device, capacidades_total.data(), capacidades_total_tam, hipMemcpyHostToDevice);

        // NUM ESTACIONES
        int num_estaciones_device = capacidades[0].size();

        // FACTOR STRESS
        float factor_stress_device = config.getFactorStress();

        // POR SEGURIDAD
        int por_seguridad_device = static_cast<int>(config.getPorSeguridad());

        // TOTAL BICIS
        int totalbicis_device = config.getTotalBicis();

        // VALOR KMS
        float valor_kms_device = config.getValorKM();

        // FACTOR SOBRANTE
        int factor_sobrante_device = static_cast<int>(config.getFactorSobrante());

        /*
        // TENDENCIAS IN
        float * tendenciasIn_device;
        int tendenciasIn_tam = tendenciasIn.size() * sizeof(float);
        hipMalloc(&tendenciasIn_device, tendenciasIn_tam);
        hipMemcpy(tendenciasIn_device, tendenciasIn.data(), tendenciasIn_tam, hipMemcpyHostToDevice);

        // TENDENCIAS OUT
        float * tendenciasOut_device;
        int tendenciasOut_tam = tendenciasOut.size() * sizeof(float);
        hipMalloc(&tendenciasOut_device, tendenciasOut_tam);
        hipMemcpy(tendenciasOut_device, tendenciasOut.data(), tendenciasOut_tam, hipMemcpyHostToDevice);
        */
        // PLAZAS SOBRANTES
        float * plazas_sobrantes_device;
        int plazas_sobrantes_tam = plazasSobrantes.size() * sizeof(float);
        hipMalloc(&plazas_sobrantes_device, plazas_sobrantes_tam);
        hipMemcpy(plazas_sobrantes_device, plazasSobrantes.data(), plazas_sobrantes_tam, hipMemcpyHostToDevice);

        // COSTE KMS EXTRA  
        float * costeKmsExtra_device;
        int costeKmsExtra_tam = costeKmsExtra.size() * sizeof(float);
        hipMalloc(&costeKmsExtra_device, costeKmsExtra_tam);
        hipMemcpy(costeKmsExtra_device, costeKmsExtra.data(), costeKmsExtra_tam, hipMemcpyHostToDevice);

        // COSTE KMS TENDENCIA
        float * costeKmsTendencia_device;
        int costeKmsTendencia_tam = costeKmsTendencia.size() * sizeof(float);
        hipMalloc(&costeKmsTendencia_device, costeKmsTendencia_tam);
        hipMemcpy(costeKmsTendencia_device, costeKmsTendencia.data(), costeKmsTendencia_tam, hipMemcpyHostToDevice);

        // idEstaciones
        int idEstaciones_device = Poblacion[0].getNumEstaciones();

        // USUARIOS PERDIDOS
        int * usuariosPerdidos_device;
        int usuariosPerdidos_tam = usuariosPerdidos.size() * sizeof(int);
        hipMalloc(&usuariosPerdidos_device, usuariosPerdidos_tam);
        hipMemcpy(usuariosPerdidos_device, usuariosPerdidos.data(), usuariosPerdidos_tam, hipMemcpyHostToDevice);

        // ESTACIONES LLENAS
        int * estaciones_llenas_device;
        int estaciones_llenas_tam = estaciones_llenas.size() * sizeof(int);
        hipMalloc(&estaciones_llenas_device, estaciones_llenas_tam);
        hipMemcpy(estaciones_llenas_device, estaciones_llenas.data(), estaciones_llenas_tam, hipMemcpyHostToDevice);

        // ESTACIONES VACIAS
        int * estaciones_vacias_device;
        int estaciones_vacias_tam = estaciones_vacias.size() * sizeof(int);
        hipMalloc(&estaciones_vacias_device, estaciones_vacias_tam);
        hipMemcpy(estaciones_vacias_device, estaciones_vacias.data(), estaciones_vacias_tam, hipMemcpyHostToDevice);
        int num_bloques = ceil(Poblacion.size() / 32);

        cuda_evaluacion<<<num_bloques, 32>>>(capacidades_device, filas_capacidades, columnas_capacidades,
                                            fitness_device, num_estaciones_device, deltas_device, factor_stress_device,
                                            por_seguridad_device, totalbicis_device, capacidades_total_device, valor_kms_device,
                                            tendencias_device, /*tendenciasIn_device, tendenciasOut_device,*/ usuariosPerdidos_device,
                                            factor_sobrante_device, cercanias_device, cercanias_km_device, columnas_cercanias_km,
                                            costeKmsExtra_device, costeKmsTendencia_device, plazas_sobrantes_device,
                                            estaciones_llenas_device, estaciones_vacias_device, filas_deltas, columnas_deltas,
                                            idEstaciones_device, columnas_cercanias, columnas_tendencias);

        hipDeviceSynchronize();

        // Copia de valores de la GPU a la CPU
        hipMemcpy(fitness.data(), fitness_device, fitness_tam, hipMemcpyDeviceToHost);
        hipMemcpy(plazasSobrantes.data(), plazas_sobrantes_device, plazas_sobrantes_tam, hipMemcpyDeviceToHost);
        hipMemcpy(costeKmsExtra.data(), costeKmsExtra_device, costeKmsExtra_tam, hipMemcpyDeviceToHost);
        hipMemcpy(costeKmsTendencia.data(), costeKmsTendencia_device, costeKmsTendencia_tam, hipMemcpyDeviceToHost);
        hipMemcpy(usuariosPerdidos.data(), usuariosPerdidos_device, usuariosPerdidos_tam, hipMemcpyDeviceToHost);
        hipMemcpy(estaciones_llenas.data(), estaciones_llenas_device, estaciones_llenas_tam, hipMemcpyDeviceToHost);
        hipMemcpy(estaciones_vacias.data(), estaciones_vacias_device, estaciones_vacias_tam, hipMemcpyDeviceToHost);
    
        // Liberamos la memoria en la GPU
        hipFree(capacidades_device);
        hipFree(fitness_device);
        hipFree(capacidades_total_device);
        //hipFree(tendenciasIn_device);
        //hipFree(tendenciasOut_device);
        hipFree(plazas_sobrantes_device);
        hipFree(costeKmsExtra_device);
        hipFree(costeKmsTendencia_device);
        hipFree(usuariosPerdidos_device);
        hipFree(estaciones_llenas_device);
        hipFree(estaciones_vacias_device);
    }
    catch(exception & e){
        cerr << "Error en la evaluación de la población: " << e.what() << endl;
        guardar_error(config);
    }
    try{
        for(int i = 0; i < fitness.size(); i++){
            Poblacion[i].setFitness(fitness[i]);
            Poblacion[i].setSobrante(plazasSobrantes[i]);
            Poblacion[i].setCosteKmExtra(costeKmsExtra[i]);
            Poblacion[i].setCosteKmsTendencia(costeKmsTendencia[i]);
            Poblacion[i].setUsuariosPerdidos(usuariosPerdidos[i]);
            Poblacion[i].setEstacionesLlenas(estaciones_llenas[i]);
            Poblacion[i].setEstacionesVacias(estaciones_vacias[i]);    
        }
        config.setEvaluaciones(config.getEvaluaciones() + fitness.size());
        config.setNLLamadasEvaluacion(config.getNLLamadasEvaluacion() + 1);
    }
    catch(exception & e){
        cerr << "Error en la actualización de la población: " << e.what() << endl;
        guardar_error(config);
    }
}